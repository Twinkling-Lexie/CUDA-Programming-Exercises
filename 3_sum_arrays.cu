#include "hip/hip_runtime.h"
/*
* https://github.com/Tony-Tan/CUDA_Freshman
* 3_sum_arrays.cu
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "freshman.h"

void sumArrays(float * a, float * b, float * res, const int size){
    for(int i=0; i<size; i+=4){
        res[i] = a[i] + b[i];
        res[i+1] = a[i+1] + b[i+1];
        res[i+2] = a[i+2] + b[i+2];
        res[i+3] = a[i+3] + b[i+3];
    }
}

__global__ void sumArraysGPU(float * a, float * b, float * res){
    int i = threadIdx.x;
    res[i] = a[i] + b[i];
}

int main(int argc, char **argv){
    int dev = 0;
    hipSetDevice(dev);

    int nElem = 32;
    printf("Vector size:%d\n", nElem);
    int nByte = sizeof(float)*nElem;
    float *a_h   = (float*)malloc(nByte);
    float *b_h   = (float*)malloc(nByte);
    float *res_h = (float*)malloc(nByte);
    float *res_from_gpu_h = (float*)malloc(nByte);
    memset(res_h, 0, nByte);
    memset(res_from_gpu_h, 0, nByte);

    float *a_d, *b_d, *res_d;
    CHECK(hipMemcpy())
}


