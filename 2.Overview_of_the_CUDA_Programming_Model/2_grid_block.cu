/*
* 2_grid_block.cu
* nvcc -o a 2_grid_block.cu && ./a && rm a
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv){
    int nElem = 1024;
    dim3 block(1024);
    dim3 grid((nElem-1)/block.x+1);
    printf("while block.x = 1024 :\n");
    printf("grid.x = %d, grid.y = %d, grid.z = %d\n", grid.x, grid.y, grid.z);
    printf("block.x= %d, block.y= %d, block.z= %d\n", block.x, block.y, block.z);
    printf("\n");

    block.x = 512;
    block.x = (nElem-1)/block.x + 1;
    printf("while block.x = 512 :\n");
    printf("grid.x = %d, grid.y = %d, grid.z = %d\n", grid.x, grid.y, grid.z);
    printf("block.x= %d, block.y= %d, block.z= %d\n", block.x, block.y, block.z);
    printf("\n");

    block.x = 256;
    block.x = (nElem-1)/block.x + 1;
    printf("while block.x = 256 :\n");
    printf("grid.x = %d, grid.y = %d, grid.z = %d\n", grid.x, grid.y, grid.z);
    printf("block.x= %d, block.y= %d, block.z= %d\n", block.x, block.y, block.z);
    printf("\n");

    block.x = 128;
    block.x = (nElem-1)/block.x + 1;
    printf("while block.x = 128 :\n");
    printf("grid.x = %d, grid.y = %d, grid.z = %d\n", grid.x, grid.y, grid.z);
    printf("block.x= %d, block.y= %d, block.z= %d\n", block.x, block.y, block.z);
    printf("\n");

    hipDeviceReset();
    return 0;
}