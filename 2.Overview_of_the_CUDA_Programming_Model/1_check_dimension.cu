/*
* 1_check_dimension.cu
* nvcc -o a 1_check_dimension.cu && ./a && rm a
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void checkIndex(void){
    printf("threadIdex:(%d,%d,%d) blockIdx:(%d,%d,%d) blockDim:(%d,%d,%d) gridDim(%d,%d,%d)\n",
        threadIdx.x,threadIdx.y,threadIdx.z,
        blockIdx.x,blockIdx.y,blockIdx.z,
        blockDim.x, blockDim.y, blockDim.z,
        gridDim.x,  gridDim.y,  gridDim.z);
    return;
}

int main(int argc, char **argv){
    int nElem = 6;
    dim3 block(3);
    dim3 grid((nElem+block.x - 1)/block.x);
    /*
    * 计算gridDim的方式(nElem+block.x-1)/block.x是为了确保能够覆盖所有的元素，
         即使这些元素不能完全被块的大小整除。
    * (nElem + block.x - 1): 这一步是为了确保即使nElem不能被block.x整除，
         也能计算出正确的网格尺寸。这里减去1是为了让计算更加精确，
         因为如果你直接加上block.x，可能会多算一个块。
    * block.x: 这一步是将经过调整后的总元素数量除以每个块中的线程数量，
         从而得出需要多少个块来处理所有的元素。
    *公式的优点
        1. 确保所有元素都被处理:
           即使元素总数不是块大小的倍数，也能确保所有元素都被分配到块中。
        2. 避免多余的块:
           如果元素总数刚好是块大小的倍数，则不会产生多余的块。
    */

    printf("grid.x=%d, grid.y=%d, grid.z=%d\n",grid.x,grid.y,grid.z);
    printf("block.x=%d, block.y=%d, block.z=%d\n",block.x,block.y,block.z);
    checkIndex<<<grid,block>>>();
    hipDeviceReset();
    return 0;
}