/*
    hello_world.cu
    nvcc -o a hello_world.cu&& ./a && rm a
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

// Each of these ten threads will print the text once.
__global__ void hello_world(void){
    printf("printed by %d : GPU: Hello world!\n",threadIdx.x);
    // printf("GPU: Hello world!\n");
}

int main(){
    // Here "CPU: Hello world!\n" is a string printed by CPU(Host).
    printf("CPU: Hello world!\n");
    hello_world<<<1,10>>>();
    hipDeviceReset();
    return 0;
}